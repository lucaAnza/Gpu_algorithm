/* 

Programm to the test if GPU works

*/







#include <hip/hip_runtime.h>
#include<stdio.h>



// Funzione esegue la somma di n elementi
__global__ void test_gpu(int *arr , int n , int* sum_gpu ){
	
    int id = threadIdx.x;
	if(id<n){
		int add_factor = 12;
        arr[id] = add_factor;
		atomicAdd( sum_gpu , add_factor);
	}

}
		
int main(){
	

	int n = 5; // number of threads
	int *GpuArr;
	int CpuArr[n];
	int* sum_gpu;
	int sum_cpu = 0;
	hipMalloc(&GpuArr , sizeof(int) * n);
	hipMalloc(&sum_gpu , sizeof(int) * 1);

	test_gpu<<<1,n>>>(GpuArr , n , sum_gpu);
	
	hipMemcpy(CpuArr , GpuArr , sizeof(int) * n , hipMemcpyDeviceToHost );
	hipMemcpy(&sum_cpu , sum_gpu , sizeof(int) * 1 , hipMemcpyDeviceToHost );
	hipDeviceSynchronize();

	//stampa array finale
	printf("sum_gpu = %d\n" , sum_cpu);
	
	for ( int i=0 ; i<n ; i++){
		printf("%d\n" , CpuArr[i]);
	}

	return 0;

}