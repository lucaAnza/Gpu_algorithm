#include<stdio.h>
#include<hip/hip_runtime.h>

#define BLOCKSIZE 10
__global__ void dkernel(){
	printf("ID : %d\n" , threadIdx.x);
}

int main() {
	dkernel<<<1,BLOCKSIZE>>>();
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(error));
	}
}
