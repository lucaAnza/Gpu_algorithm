#include "hip/hip_runtime.h"
/**
* This file is part of Cuda accelerated ORB-SLAM project by Filippo Muzzini, Nicola Capodieci, Roberto Cavicchioli and Benjamin Rouxel.
 * Implemented by Filippo Muzzini.
 *
 * Based on ORB-SLAM2 (Raúl Mur-Artal, José M.M. Montiel and Juan D. Tardós) and ORB-SLAM3 (Carlos Campos, Richard Elvira, Juan J. Gómez Rodríguez, José M.M. Montiel and Juan D. Tardós)
 *
 * Project under GPLv3 Licence
*
*/

#include <hip/hip_runtime.h>
#include <opencv2/core/hal/interface.h>
#include <stdio.h>
#include "gpu_stereoMatches.h"

#include <vector>
#include <opencv2/core/core.hpp>

//Allocazione memoria costante in Gpu                      
__constant__  float minZ_gpu;   
__constant__  float minD_gpu;                           
__constant__  float maxD_gpu;  
__constant__  int TH_HIGH_gpu;


__global__ void cuda_test(size_t* vRowIndices_gpu , cv::KeyPoint *mvKeys_gpu , float* mDescriptors_gpu , float *mDescriptorsRight_gpu , float *mvInvScaleFactors_gpu  , float *mvScaleFactors_gpu 
                        , size_t *size_refer_gpu ) {
    
    int id = threadIdx.x;
    int b_id = blockIdx.x;

    if(id < size_refer_gpu[b_id]){
        printf("riga %d , elemento[%d] = " , b_id , id , -1 );
    }

    printf("\n\n\n");    

}


void gpu_stereoMatches(std::vector<std::vector<size_t>> vRowIndices , std::vector<cv::KeyPoint> mvKeys , float minZ , float minD , float maxD , int TH_HIGH , cv::Mat mDescriptors , cv::Mat mDescriptorsRight , 
                      std::vector<float> mvInvScaleFactors , std::vector<float> mvScaleFactors , std::vector<size_t> size_refer ){

    cv::KeyPoint *mvKeys_gpu;
    float *mvInvScaleFactors_gpu;
    float *mDescriptorsRight_gpu;
    float *mDescriptors_gpu;
    float *mvScaleFactors_gpu;
    size_t *size_refer_gpu;
    size_t *vRowIndices_gpu;
    int num_elements_left = mDescriptors.total();
    int num_elements_right = mDescriptorsRight.total();
    unsigned total_element=0;
    unsigned nRows = vRowIndices.size();

    
    // Copia parametri input in memoria costante
    hipMemcpyToSymbol(HIP_SYMBOL(minZ_gpu), &minZ, 1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(minD_gpu), &minD, 1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(maxD_gpu), &maxD, 1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(TH_HIGH_gpu), &TH_HIGH, 1 * sizeof(int));

    //Allocazione memoria per array dinamici
    hipMalloc(&mvKeys_gpu , sizeof(cv::KeyPoint) * mvKeys.size() );
    hipMemcpy(mvKeys_gpu, mvKeys.data(), sizeof(cv::KeyPoint) * mvKeys.size(), hipMemcpyHostToDevice); 
    hipMalloc(&mDescriptors_gpu, num_elements_left * sizeof(float));
    hipMemcpy(mDescriptors_gpu, (float*)mDescriptors.data, num_elements_left * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&mDescriptorsRight_gpu, num_elements_right * sizeof(float));
    hipMemcpy(mDescriptorsRight_gpu, (float*)mDescriptorsRight.data, num_elements_right * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&mvInvScaleFactors_gpu , sizeof(float) * mvInvScaleFactors.size() );
    hipMemcpy(mvInvScaleFactors_gpu, mvInvScaleFactors.data(), sizeof(float) * mvInvScaleFactors.size(), hipMemcpyHostToDevice); 
    hipMalloc(&mvScaleFactors_gpu , sizeof(float) * mvScaleFactors.size() );
    hipMemcpy(mvScaleFactors_gpu, mvScaleFactors.data(), sizeof(float) * mvScaleFactors.size(), hipMemcpyHostToDevice);
    hipMalloc(&size_refer_gpu , sizeof(size_t) * size_refer.size() );
    hipMemcpy(size_refer_gpu, size_refer.data(), sizeof(size_t) * size_refer.size(), hipMemcpyHostToDevice); 
    //TODO -> Evitare di fare questo ciclo e di allocare vRowIndices_temp (spreco di memoria e tempo)
    std::vector<size_t> vRowIndices_temp;
    for(int i=0 ; i<vRowIndices.size() ; i++){
        for(int j=0; j<vRowIndices[j].size() ; j++){
            total_element++;
            vRowIndices_temp.push_back(vRowIndices[i][j]);
        }
    }
    hipMalloc(&vRowIndices_gpu , sizeof(size_t) * total_element );
    hipMemcpy(vRowIndices_gpu, vRowIndices.data(), sizeof(size_t) * total_element, hipMemcpyHostToDevice); 

         
    
    printf("Sto per lanciare il test della GPU by Luca Anzaldi: \n");
    //Ogni blocco rappresenta una riga di VrowIndices e ogni thread le varie colonne
    cuda_test<<<nRows,200>>>(vRowIndices_gpu , mvKeys_gpu , mDescriptors_gpu ,mDescriptorsRight_gpu , mvInvScaleFactors_gpu, mvScaleFactors_gpu , size_refer_gpu );
    hipDeviceSynchronize();

    //Deallocazione della memoria
    hipFree(mvKeys_gpu);
    hipFree(mvInvScaleFactors_gpu);
    hipFree(mDescriptorsRight_gpu);
    hipFree(mDescriptors_gpu);
    hipFree(mvScaleFactors_gpu);
    hipFree(size_refer_gpu);
    hipFree(vRowIndices_gpu);
}



/*
void gpu_stereoMatches(std::vector<cv::KeyPoint> mvKeys , float minZ , float minD , float maxD , int TH_HIGH , cv::Mat mDescriptorsRight , 
                        vector<float> mvInvScaleFactors , ORBextractor* mpORBextractorLeft , vector<float> mvScaleFactors ){
*/
