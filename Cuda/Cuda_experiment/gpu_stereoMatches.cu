#include "hip/hip_runtime.h"
/**
* This file is part of Cuda accelerated ORB-SLAM project by Filippo Muzzini, Nicola Capodieci, Roberto Cavicchioli and Benjamin Rouxel.
 * Implemented by Filippo Muzzini.
 *
 * Based on ORB-SLAM2 (Raúl Mur-Artal, José M.M. Montiel and Juan D. Tardós) and ORB-SLAM3 (Carlos Campos, Richard Elvira, Juan J. Gómez Rodríguez, José M.M. Montiel and Juan D. Tardós)
 *
 * Project under GPLv3 Licence
*
*/

#include <hip/hip_runtime.h>
#include <opencv2/core/hal/interface.h>
#include <stdio.h>
#include "gpu_stereoMatches.h"

#include <vector>
#include <opencv2/core/core.hpp>


//Allocazione memoria costante in Gpu                      
__constant__  float minZ_gpu;   
__constant__  float minD_gpu;                           
__constant__  float maxD_gpu;  
__constant__  int TH_HIGH_gpu;


__global__ void cuda_test(cv::KeyPoint *mvKeys_gpu) {
    
    int temp = threadIdx.x;

    printf("%f , %f , %f \n" , minZ_gpu , minD_gpu , maxD_gpu );
    printf("punto prova di mvKeys[0] : %f \n\n\n" , mvKeys_gpu->pt.x);

}


void gpu_stereoMatches(std::vector<cv::KeyPoint> mvKeys , float minZ , float minD , float maxD  ){

    cv::KeyPoint *mvKeys_gpu;
    
    // Copia parametri input in memoria costante
    hipMemcpyToSymbol(HIP_SYMBOL(minZ_gpu), &minZ, 1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(minD_gpu), &minD, 1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(maxD_gpu), &maxD, 1 * sizeof(float));

    //Allocazione memoria per array dinamici
    hipMalloc(&mvKeys_gpu , sizeof(cv::KeyPoint) * mvKeys.size() );
    hipMemcpy(mvKeys_gpu, mvKeys.data(), sizeof(cv::KeyPoint) * mvKeys.size(), hipMemcpyHostToDevice); 


    printf("Sto per lanciare il test della GPU by Luca Anzaldi: \n");
    cuda_test<<<1,1>>>(mvKeys_gpu);
}



/*

void gpu_stereoMatches(std::vector<cv::KeyPoint> mvKeys , float minZ , float minD , float maxD , int TH_HIGH , cv::Mat mDescriptorsRight , 
                        vector<float> mvInvScaleFactors , ORBextractor* mpORBextractorLeft , vector<float> mvScaleFactors ){

*/
