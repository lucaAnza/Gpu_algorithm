/* Optimization-1 
-> Attualmente il programma cerca di moltiplicare ogni elemento di una riga per l'indice della riga+1 in parallelo.

-> Per compilare: "nvcc -lcublas opt1_simulation.cu"
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include <hipblas.h>

using namespace std;


// Funzione che moltiplica ogni elemento di una riga per l'indice di riga + 1
__global__ void mul_element_for_lineIndex(float *arr , int const *index_refer,  int n){
	int id = threadIdx.x;
	
    arr[id] = arr[id] * index_refer[id];
}
	
	
int main(){

    const int nRows = 4;   // Numero di righe dell'immagine sinistra
    //vector<vector<size_t> > vRowIndices(nRows,vector<size_t>());   // Crea una matrice con nRows vettori 

    // Creazione matrice VrowIndices
    vector<vector<size_t> > vRowIndices = {
        {3, 7, 15},
        {5, 6, 12, 22},
        {2, 8, 9},
        {6, 7}
    };
    //Ampliamento matrice VrowIndices
    for(int i=0; i<nRows; i++)
        vRowIndices[i].reserve(200);   

    // Stampa di VrowIndices 
    cout<<"\nVRowIndices : "<<endl;
    int n=0;
    for(int i=0 ; i<nRows ; i++){
        for(int j=0 ; j<vRowIndices[i].size() ; j++){
            cout<<vRowIndices[i][j]<<" ";
            n++;
        }
        cout<<endl;
    }
    
    // Allocazione vettore su Gpu e Cpu
    float *GpuArr;
    float *CpuArr = new float[n];
    int *index_refer_gpu;
    int *index_refer_cpu = new int[n];
	hipMalloc(&GpuArr , sizeof(float) * n );  
    hipMalloc(&index_refer_gpu , sizeof(int) * n );  


    // Init vettore su Gpu
    size_t c = 0;
    for(int i = 0; i < nRows; i++) {
        for(int j = 0; j < vRowIndices[i].size(); j++) {
            // Copia il valore dal vettore vRowIndices al vettore GpuArr
            float temp = (float) vRowIndices[i][j];
            hipMemcpy(&GpuArr[c], &temp , sizeof(float), hipMemcpyHostToDevice);
            index_refer_cpu[c] = i+1;
            c++;
        }
    }
    hipMemcpy(index_refer_gpu, index_refer_cpu, sizeof(int) * n, hipMemcpyHostToDevice);


    // Chiamata funzione + spostamento gpu -> cpu
    mul_element_for_lineIndex<<<1,n>>>(GpuArr , index_refer_gpu , n);
    hipMemcpy(CpuArr , GpuArr , sizeof(float) * n , hipMemcpyDeviceToHost );
    hipMemcpy(index_refer_cpu , index_refer_gpu , sizeof(int) * n , hipMemcpyDeviceToHost );

    // Stampa di GpuArr
    cout<<"\nVettore su Cpu : "<<endl;
    for(int i=0 ; i<n ; i++){
        cout<<CpuArr[i]<<" ";
    }
    // Stampa di Index refer
    cout<<"\nIndex refer: "<<endl;
    for(int i=0 ; i<n ; i++){
        cout<<index_refer_cpu[i]<<" ";
    }
    cout<<endl;


    // Calcolo del minimo per ogni array
    int offset = 0;
    int result;
    int size;
    int *minium_array = new int[nRows];
    hipblasHandle_t handle;
    for(int i=0 ; i<nRows ; i++){
        hipblasCreate(&handle);
        size = vRowIndices[i].size();
        hipblasIsamin(handle, size , GpuArr + offset, 1, &result);
        offset += size;
        std::cout << "\nMinimum element of line "<<i<<"is at index: " << result << std::endl;
        minium_array[i] = result;
        hipblasDestroy(handle);
    }

    // Sort of minium_array

	hipDeviceSynchronize();
    
	return 0;

}