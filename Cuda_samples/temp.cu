
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void dkernel(){

    if( threadIdx.x == 0 && blockIdx.x == 0 && 
        threadIdx.y == 0 && blockIdx.y == 0 && 
        threadIdx.z == 0 && blockIdx.z == 0 ) {
            printf("%d %d %d %d %d %d\n" , gridDim.x , gridDim.y , gridDim.z , blockDim.x , blockDim.y , blockDim.z );
    }

}

int main(){
    dim3 grid(2,3,4);  //set size of each grid
    dim3 block(5,6,7);   // set size of each block
    dkernel<<<grid,block>>>();
    hipDeviceSynchronize();
}